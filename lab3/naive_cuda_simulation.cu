#include "hip/hip_runtime.h"
#include "naive_cuda_simulation.cuh"
#include "physics/gravitation.h"
#include "physics/mechanics.h"
#include "simulation/constants.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_wrappers.cuh"


// DEVICE FRIENDLY HILFSMETHODEN:


__constant__ double gravitational_constant_device = 6.67430e-11; // (m^3)/(kg*s^2)


__host__ __device__ inline
double compute_gravitational_force_device(double mass_1, double mass_2, double distance) {
    return gravitational_constant_device * ((mass_1 * mass_2) / (distance * distance));
}


constexpr double epoch_time_in_seconds_device = 2.628e+6; 




void NaiveCudaSimulation::allocate_device_memory(Universe& universe, void** d_weights, void** d_forces, void** d_velocities, void** d_positions){

 size_t num_bodies = universe.num_bodies;

//-> num bodies times double
hipMalloc(d_weights, num_bodies * sizeof(double));

//-> VECTOR 2D IN PREV MODULES IS NOW DOUBLE2
    hipMalloc(d_forces, num_bodies * sizeof(double2));
    hipMalloc(d_velocities, num_bodies * sizeof(double2));
    hipMalloc(d_positions, num_bodies * sizeof(double2));


}

void NaiveCudaSimulation::free_device_memory(void** d_weights, void** d_forces, void** d_velocities, void** d_positions){

hipFree(*d_weights);
hipFree(*d_forces);
hipFree(*d_velocities);
hipFree(*d_positions);

*d_weights = nullptr;
    *d_forces = nullptr;
    *d_velocities = nullptr;
    *d_positions = nullptr;


}


/*
implementation decision
-> no explicit performance criteria but instead of copying entire vectors, recieving constant references
better
-> check later if any problems

*/
std::vector<double2> vector2d_to_double2_translator(const std::vector<Vector2d<double>>& vector2d){

    size_t vector_size = vector2d.size(); //ideally num bodies if debug needed
    std::vector<double2> vector_to_return(vector_size);

    for (size_t i = 0; i < vector_size; i++) {
        //0 = x 1= y
        vector_to_return[i] = make_double2(vector2d[i][0], vector2d[i][1]);
    }

    return vector_to_return;
}


void NaiveCudaSimulation::copy_data_to_device(Universe& universe, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){

/*

-> weight in plain double type

// DOES IT WORK???
     expected = an pointer stating the start of the array, like c type
     .data() returns a pointer for c++ vector types
     another implementation (doesnt match with verbindliche anforderung but implement if error)
    for loop for cudamemcpy or c type of arrays


*/
size_t num_bodies = universe.num_bodies;



//weights are in plain dobule vector type, no translation needed, loading directly
hipMemcpy(d_weights, universe.weights.data(), num_bodies * sizeof(double), hipMemcpyHostToDevice);

 std::vector<double2> forces = vector2d_to_double2_translator(universe.forces);
    std::vector<double2> velocities = vector2d_to_double2_translator(universe.velocities);
    std::vector<double2> positions = vector2d_to_double2_translator(universe.positions);



hipMemcpy(d_forces, forces.data(), num_bodies * sizeof(double2), hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, velocities.data(), num_bodies * sizeof(double2), hipMemcpyHostToDevice);
    hipMemcpy(d_positions, positions.data(), num_bodies * sizeof(double2), hipMemcpyHostToDevice);

}

std::vector<Vector2d<double>> double2_to_vector2d_translator(const std::vector<double2>& cuda_vector){

    size_t vector_size = cuda_vector.size(); //ideally num bodies if debug needed
    std::vector<Vector2d<double>> vector_to_return(vector_size);

    for (size_t i = 0; i < vector_size; ++i) {
        //0 = x 1= y
        vector_to_return[i] = Vector2d{cuda_vector[i].x,cuda_vector[i].y};
    }

    return vector_to_return;
}


/*
implementation decision/gedankengang:
    -> saving values into universe object, as no explicit host variables are given.


*/
void NaiveCudaSimulation::copy_data_from_device(Universe& universe, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){
 
     size_t num_bodies = universe.num_bodies;

     hipMemcpy(universe.weights.data(), d_weights, num_bodies * sizeof(double), hipMemcpyDeviceToHost);


    //pre translation data types
 std::vector<double2> forces(num_bodies);
    std::vector<double2> velocities(num_bodies);
    std::vector<double2> positions(num_bodies);

 
 
 hipMemcpy(forces.data(), d_forces, num_bodies * sizeof(double2), hipMemcpyDeviceToHost);
    hipMemcpy(velocities.data(), d_velocities, num_bodies * sizeof(double2), hipMemcpyDeviceToHost);
    hipMemcpy(positions.data(), d_positions, num_bodies * sizeof(double2), hipMemcpyDeviceToHost);


//setting universe variables to translated data types

universe.forces = double2_to_vector2d_translator(forces);
    universe.velocities = double2_to_vector2d_translator(velocities);
    universe.positions = double2_to_vector2d_translator(positions);


}



/*
> implementation decision: just convert one iteration of naive sequential
to cuda kernel + add schelifenbedingung as an if.


ERROR: GRAVITIONAL FORCE IS NOT WELL DEFINED IN GPU USE! -> changed


 */
__global__
void calculate_forces_kernel(std::uint32_t num_bodies, double2* d_positions, double* d_weights, double2* d_forces){

//at the same time himmelskörper index, as
    //body_id_x in sequential function
int thread_himmelskorper_index = blockIdx.x * blockDim.x + threadIdx.x;

//last legal move= num_bodies -1, right?
if (thread_himmelskorper_index < num_bodies){

    double2 body_position = d_positions[thread_himmelskorper_index];
    double body_mass = d_weights[thread_himmelskorper_index];

     double2 applied_force_vector = make_double2(0.0, 0.0);

     for (int distant_body_idx = 0; distant_body_idx < num_bodies; ++distant_body_idx) {

        //skip current korper:


        if (thread_himmelskorper_index == distant_body_idx){
            continue;
        }


          double2 distant_body_position = d_positions[distant_body_idx];
        double distant_body_mass = d_weights[distant_body_idx];

        double2 direction_vector = make_double2(
                distant_body_position.x - body_position.x,
                distant_body_position.y - body_position.y
            );



        double distance = sqrt(pow(direction_vector.x, 2) + pow(direction_vector.y, 2));

        double force_magnitude = compute_gravitational_force_device(body_mass, distant_body_mass, distance);


        double2 force_vector;
        force_vector.x = (direction_vector.x / distance) * force_magnitude;
        force_vector.y = (direction_vector.y / distance) * force_magnitude;


        applied_force_vector.x += force_vector.x;
        applied_force_vector.y += force_vector.y;

     }//end of for for distant bodies

    //save results
     d_forces[thread_himmelskorper_index] = applied_force_vector;

}//end of schleifenbedingung checker if

}


/*
implementation decision: how to choose block/grid/tile etc. sizes?
    in vorlesung = tile_width was an pre-defined macro and we haven't talked about it much

    reseaarch: a sweet spoot between 128-512 (must be dividable by warp size 32)
    -> occupancy api: too complez, we have no performance criteria, turn back if comes in upcoming aufgaben

    -> naive and sequential shouldnt return different results -> should we do one by one???? TEST!!!!


    -> DO WE NEED POINTER CASTING???

 */
void NaiveCudaSimulation::calculate_forces(Universe& universe, void* d_positions, void* d_weights, void* d_forces){


    //each block would calculate 256 results, 256 bodies
     int thread_amount_pro_block = 256;

     //how many blocks are needed given result array size:
    int amount_blocks = (universe.num_bodies + thread_amount_pro_block - 1) / thread_amount_pro_block;



     calculate_forces_kernel<<<amount_blocks, thread_amount_pro_block>>>(
        universe.num_bodies, 
        (double2*)d_positions, 
        (double*)d_weights,     
        (double2*)d_forces
    );

        hipDeviceSynchronize();


}


__device__
double2 calculate_acceleration_in_double2(double2 applied_force, double mass) {
    
    // calculate acceleration 
    // a = F / m
    return make_double2(applied_force.x / mass, applied_force.y / mass);
}


__device__
double2 calculate_velocity_in_double2(double2 base_velocity, double2 acceleration, double time_in_seconds) {
    
    // v = v0 + a * t
    return make_double2(
        base_velocity.x + acceleration.x * time_in_seconds,
        base_velocity.y + acceleration.y * time_in_seconds
    );
}


/*

BIG TODO: DO WE INJECT "static const double epoch_in_seconds = 2.628e+6;" HERE?
CHECK IN A ENVIROMENT WITH MORE STABLE INTELLISENSE!!!!!!!!!!
-> currently white!!!!!!!!!!! -> added constant.h as header, do i miss sth?


*/
__global__
void calculate_velocities_kernel(std::uint32_t num_bodies, double2* d_forces, double* d_weights, double2* d_velocities){

    int thread_himmelskorper_index = blockIdx.x * blockDim.x + threadIdx.x;
    

    if(thread_himmelskorper_index< num_bodies){

    double2 force = d_forces[thread_himmelskorper_index];
    double mass = d_weights[thread_himmelskorper_index];

    
    double2 acceleration = calculate_acceleration_in_double2(force, mass);
    d_velocities[thread_himmelskorper_index] = calculate_velocity_in_double2(d_velocities[thread_himmelskorper_index], acceleration, epoch_time_in_seconds_device);



    }//end of schleifenbedingung if 


}

void NaiveCudaSimulation::calculate_velocities(Universe& universe, void* d_forces, void* d_weights, void* d_velocities){

  //each block would calculate 256 results, 256 bodies
     int thread_amount_pro_block = 256;

     //how many blocks are needed given result array size:
    int amount_blocks = (universe.num_bodies + thread_amount_pro_block - 1) / thread_amount_pro_block;


 calculate_velocities_kernel<<<amount_blocks , thread_amount_pro_block>>>(
        universe.num_bodies, 
        (double2*)d_forces,  // Use the passed argument
        (double*)d_weights,  // Use the passed argument
        (double2*)d_velocities  // Use the passed argument
    );

    hipDeviceSynchronize();




}

__global__
void calculate_positions_kernel(std::uint32_t num_bodies, double2* d_velocities, double2* d_positions){

int thread_himmelskorper_index = blockIdx.x * blockDim.x + threadIdx.x;


  if(thread_himmelskorper_index< num_bodies){


    double2 velocity = d_velocities[thread_himmelskorper_index];
    double2 position = d_positions[thread_himmelskorper_index];

// calculate movement
        // s = v * t
     double2 movement;
    movement.x = velocity.x * epoch_time_in_seconds_device;
    movement.y = velocity.y * epoch_time_in_seconds_device;

// calculate new position
        // p` = p0 + s 
     double2 new_position;
    new_position.x = position.x + movement.x;
    new_position.y = position.y + movement.y;


     d_positions[thread_himmelskorper_index] = new_position;


  }//end of schleifenbedingung if


}

void NaiveCudaSimulation::calculate_positions(Universe& universe, void* d_velocities, void* d_positions){


//each block would calculate 256 results, 256 bodies
     int thread_amount_pro_block = 256;

     //how many blocks are needed given result array size:
    int amount_blocks = (universe.num_bodies + thread_amount_pro_block - 1) / thread_amount_pro_block;


 calculate_positions_kernel<<<amount_blocks, thread_amount_pro_block>>>(
        universe.num_bodies, 
        (double2*)d_velocities, 
        (double2*)d_positions
    );
    hipDeviceSynchronize();


}



/* implementation decisions:

ANFORDERUNG: gpu speicher allocation + freeing

sequantial workflow for ONE EPOCH

-> calculate forces
->c velocities
-> c positions
-> +1 epoch

if create intermediate plots:,
     if((universe.current_simulation_epoch % plot_intermediate_epochs) == 0){
            plotter.add_bodies_to_image(universe);
            plotter.write_and_clear();
        }


epochS : repeat given times


-> where allocating memory

-> where freeing



-> memory allocated, pointer of pointer type of pointers MUST BE CASTED TO VOID!
    -> should we make the initial pointers null pointer?
        -> i did, it wasnt like that in vorlesung example but research says better practice
            -> TODO IF ERROR: make them unitialized




 */

// this function would be called by CPU, 
void NaiveCudaSimulation::simulate_epochs(Plotter& plotter, Universe& universe, std::uint32_t num_epochs, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs){

//ASSIGN MEMORY

double* d_weights = nullptr;
double2* d_forces = nullptr;
double2* d_velocities = nullptr;
double2* d_positions = nullptr;

NaiveCudaSimulation::allocate_device_memory(universe, 
    (void**)&d_weights, 
    (void**)&d_forces, 
    (void**)&d_velocities, 
    (void**)&d_positions
    );

//COPY DATA FROM HOST TO DEVICE

NaiveCudaSimulation::copy_data_to_device(universe, d_weights, d_forces, d_velocities, d_positions);

// DO THE SIMULATIONS

 for(int i = 0; i < num_epochs; i++){
        NaiveCudaSimulation::simulate_epoch(plotter, universe, create_intermediate_plots, plot_intermediate_epochs, d_weights, d_forces, d_velocities, d_positions);
    }


//copy data back

NaiveCudaSimulation::copy_data_from_device(universe, d_weights, d_forces, d_velocities, d_positions);

//free memory
NaiveCudaSimulation::free_device_memory((void**)&d_weights, (void**)&d_forces, (void**)&d_velocities, (void**)&d_positions);

}





__global__
void get_pixels_kernel(std::uint32_t num_bodies, double2* d_positions, std::uint8_t* d_pixels, std::uint32_t plot_width, std::uint32_t plot_height, double plot_bounding_box_x_min, double plot_bounding_box_x_max, double plot_bounding_box_y_min, double plot_bounding_box_y_max){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_bodies) return;

    double2 pos = d_positions[i];

    if (pos.x >= plot_bounding_box_x_min && pos.x <= plot_bounding_box_x_max &&
        pos.y >= plot_bounding_box_y_min && pos.y <= plot_bounding_box_y_max) {
        
        int pixel_x = static_cast<int>(((pos.x - plot_bounding_box_x_min) / 
                      (plot_bounding_box_x_max - plot_bounding_box_x_min)) * (plot_width - 1));
        int pixel_y = static_cast<int>(((pos.y - plot_bounding_box_y_min) / 
                      (plot_bounding_box_y_max - plot_bounding_box_y_min)) * (plot_height - 1));

        if (pixel_x >= 0 && pixel_x < plot_width && pixel_y >= 0 && pixel_y < plot_height) {
            int pixel_index = pixel_y * plot_width + pixel_x;
            d_pixels[pixel_index] = 255;
        }
    }
}

std::vector<std::uint8_t> NaiveCudaSimulation::get_pixels(std::uint32_t plot_width, std::uint32_t plot_height, BoundingBox plot_bounding_box, void* d_positions, std::uint32_t num_bodies){
    std::vector<std::uint8_t> pixels(plot_width * plot_height, 0);

    uint8_t* d_pixels_void;
    size_t pixel_data_size = plot_width * plot_height * sizeof(std::uint8_t);

    parprog_cudaMalloc(reinterpret_cast<void**>(&d_pixels_void), pixel_data_size);
    hipMemset(d_pixels_void, 0, pixel_data_size);


    const int threadsPerBlock = 256;
    const int numBlocks = (plot_width * plot_height + threadsPerBlock - 1) / threadsPerBlock;

    get_pixels_kernel<<<numBlocks, threadsPerBlock>>>(
        num_bodies,                              
        static_cast<double2*>(d_positions),      
        d_pixels_void,                                
        plot_width, plot_height,                
        plot_bounding_box.x_min, plot_bounding_box.x_max,
        plot_bounding_box.y_min, plot_bounding_box.y_max
    );
    

    hipDeviceSynchronize();

    parprog_cudaMemcpy(pixels.data(), d_pixels_void, pixel_data_size, hipMemcpyDeviceToHost);

    parprog_cudaFree(d_pixels_void);

    return pixels;

}

__global__
void compress_pixels_kernel(std::uint32_t num_raw_pixels, std::uint8_t* d_raw_pixels, std::uint8_t* d_compressed_pixels){

}

void NaiveCudaSimulation::compress_pixels(std::vector<std::uint8_t>& raw_pixels, std::vector<std::uint8_t>& compressed_pixels){

}

void NaiveCudaSimulation::simulate_epoch(Plotter& plotter, Universe& universe, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){
    calculate_forces(universe, d_positions, d_weights, d_forces);
    calculate_velocities(universe, d_forces, d_weights, d_velocities);
    calculate_positions(universe, d_velocities, d_positions);

    universe.current_simulation_epoch++;
    if(create_intermediate_plots){
        if(universe.current_simulation_epoch % plot_intermediate_epochs == 0){
            std::vector<std::uint8_t> pixels = get_pixels(plotter.get_plot_width(), plotter.get_plot_height(), plotter.get_plot_bounding_box(), d_positions, universe.num_bodies);
            plotter.add_active_pixels_to_image(pixels);

            // This is a dummy to use compression in plotting, although not beneficial performance-wise
            // ----
            // std::vector<std::uint8_t> compressed_pixels;
            // compressed_pixels.resize(pixels.size()/8);
            // compress_pixels(pixels, compressed_pixels);
            // plotter.add_compressed_pixels_to_image(compressed_pixels);
            // ----

            plotter.write_and_clear();
        }
    }
}

void NaiveCudaSimulation::calculate_forces_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_positions, void* d_weights, void* d_forces){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_forces_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_positions, (double*) d_weights, (double2*) d_forces);
}

void NaiveCudaSimulation::calculate_velocities_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_forces, void* d_weights, void* d_velocities){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_velocities_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_forces, (double*) d_weights, (double2*) d_velocities);
}

void NaiveCudaSimulation::calculate_positions_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_velocities, void* d_positions){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_positions_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_velocities, (double2*) d_positions);
}
